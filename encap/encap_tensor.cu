#include <hip/hip_fp16.h>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>

#include "needle_tensor.cuh"
#include "needle_util.cuh"
#include "nn/function.cuh"
#include "nn/nn_module.cuh"
#include "nn/linear.cuh"
#include "init/init_basic.cuh"
#include "init/initial.hpp"

void bind_function(py::module& m) {
    m.def("arange", [](int32_t start, int32_t end, int32_t step = 1, 
                       DataType dtype = DataType::FLOAT, 
                       BackendType device = BackendType::CUDA) {
        return NdlTensor::arange(start, end, step, dtype, device);
    }, 
    py::arg("start"), 
    py::arg("end"), 
    py::arg("step") = 1, 
    py::arg("dtype") = DataType::FLOAT, 
    py::arg("device") = BackendType::CUDA);

    m.def("ones", [](std::vector<int32_t> shape,
                       DataType dtype = DataType::FLOAT, 
                       BackendType device = BackendType::CUDA) {
        return NdlTensor::ones(shape, dtype, device);
    }, 
    py::arg("shape"),
    py::arg("dtype") = DataType::FLOAT, 
    py::arg("device") = BackendType::CUDA);

    m.def("zeros", [](std::vector<int32_t> shape,
                       DataType dtype = DataType::FLOAT, 
                       BackendType device = BackendType::CUDA) {
        return NdlTensor::zeros(shape, dtype, device);
    }, 
    py::arg("shape"),
    py::arg("dtype") = DataType::FLOAT, 
    py::arg("device") = BackendType::CUDA);

    m.def("fill_val", [](std::vector<int32_t> shape,
                        float val,
                        DataType dtype = DataType::FLOAT, 
                        BackendType device = BackendType::CUDA) {
        return NdlTensor::fill_val(shape, val, dtype, device);
    }, 
    py::arg("shape"),
    py::arg("val"),
    py::arg("dtype") = DataType::FLOAT, 
    py::arg("device") = BackendType::CUDA);
}

PYBIND11_MODULE(needle, m) {
    py::enum_<DataType>(m, "DataType")
        .value("FLOAT", DataType::FLOAT)
        .value("HALF", DataType::HALF)
        .value("INT8", DataType::INT8)
        .value("INT4", DataType::INT4)
        .export_values();

    // Binding for operator+ with another Tensor
    py::enum_<BackendType>(m, "BackendType")
        .value("CPU", BackendType::CPU)
        .value("CUDA", BackendType::CUDA)
        .export_values();

    py::class_<NdlTensor> tensor_class(m, "Tensor");
    tensor_class
        .def(py::init<py::array_t<float>&, DataType, BackendType>(),
            py::arg("np_array"),
            py::arg("dtype")=DataType::FLOAT,
            py::arg("backend")=BackendType::CUDA)

        .def("to_numpy", &NdlTensor::to_numpy)
        .def("device", &NdlTensor::device)
        .def("shape", &NdlTensor::shape)
        .def("strides", &NdlTensor::strides)
        .def("matmul", &NdlTensor::matmul)


        .def("__add__", [](NdlTensor& a, NdlTensor& b) {
            return a + b;
        }, py::is_operator())

        .def("__add__", [](NdlTensor& a, float scalar) {
            return a + scalar;
        }, py::is_operator())

        // 反向的 operator+，处理 float + NdlTensor
        .def("__radd__", [](NdlTensor& a, float scalar) {
            return a + scalar;
        }, py::is_operator())

        // 使用运算符重载 @ 运算符
        .def("__matmul__", &NdlTensor::matmul)

        .def_property_readonly("shape", [](NdlTensor& self) {
            const auto& shape_vector = self.shape();  // 获取形状的 std::vector<int32_t>
            py::tuple shape_tuple(shape_vector.size());  // 创建一个与 vector 大小相同的 py::tuple
            for (size_t i = 0; i < shape_vector.size(); ++i) {
                shape_tuple[i] = shape_vector[i];  // 将 vector 的元素拷贝到 tuple 中
            }
            return shape_tuple;
        })

        .def_property_readonly("strides", [](NdlTensor& self) {
            const auto& strides_vector = self.strides();  // 获取形状的 std::vector<int32_t>
            py::tuple strides_tuple(strides_vector.size());  // 创建一个与 vector 大小相同的 py::tuple
            for (size_t i = 0; i < strides_vector.size(); ++i) {
                strides_tuple[i] = strides_vector[i];  // 将 vector 的元素拷贝到 tuple 中
            }
            return strides_tuple;
        })

        ;

    bind_function(m);

    m.attr("fp32") = DataType::FLOAT;
    m.attr("fp16") = DataType::HALF;
    m.attr("cuda") = BackendType::CUDA;
    m.attr("cpu") = BackendType::CPU;
}


